/*==========================================================================================
// 這是關於把一個影片輸入，利用 GPU 運算 ，把彩色影片轉成黑白影片

// opencv == H x W x C (BGR)

=============================================================================================*/
#include <chrono>
#include <iostream>
// nvidia cuda library  
#include "npp.h"
#include "hipblas.h"
#include "hipsparse.h"


// intel-openvino-opencv
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core.hpp>
#include <opencv2/video.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/dnn.hpp>
// DIYCUDA
#include "cuda_useful.hpp"





using namespace std;


int main(int args,char* argv[]){
	if(args != 3){
		cout << "=====================================================================================\n" ;
		cout << "openvino-init : \"C:\\Program Files (x86)\\IntelSWTools\\openvino\\bin\\setupvars.bat\"\n";
		cout << "=====================================================================================\n" ;
		cout << "--run [videoname]" << endl;
		exit(1);
	}

	// ________________  讀取 .mp4___________________________
	cv::VideoCapture cap(argv[2]);                          		 // 輸入影片檔名
	int VideoFrameH = cap.get(cv::CAP_PROP_FRAME_HEIGHT);            // 影片高
	int VideoFrameW = cap.get(cv::CAP_PROP_FRAME_WIDTH);			 // 影片寬
	double fps = cap.get(cv::CAP_PROP_FPS);    						 // 取得該影片 FPS 資訊
	int size = VideoFrameH*VideoFrameW*3;
	size_t bytes = size*sizeof(unsigned char);  // 計算需要傳輸 的 bytes = H x W x C x (uchar) 

	if(cap.isOpened()){
		// 定義指標 + 配置空間
		cv::Mat input_frame;
		cv::Mat output_frame;
		unsigned char* hptr; // CPU input_frame
		unsigned char* dptr; // GPU
		unsigned char* hptr2 = (unsigned char *)malloc(bytes); // CPU output_frame
		hipMalloc(&dptr,bytes);
		chrono::steady_clock::time_point t1;
		chrono::steady_clock::time_point t2;
		int frameIdx = 1;

		// ________________ 掃每一禎 ______________________
		while(cap.read(input_frame)){
			t1 = chrono::steady_clock::now();
			{
				hptr = input_frame.data;  //存到指標上

				hipMemcpy(dptr,hptr,bytes,hipMemcpyHostToDevice);
				//===============================================
				// do something on dptr at GPU + CUDA .....

				cuda_useful::setConstantBGR<<< 2,512 >>>(dptr,size,255,255,255);
				//=================================================
				hipMemcpy(hptr2,dptr,bytes,hipMemcpyDeviceToHost);
				hipDeviceSynchronize(); //與主程式同步
				output_frame = cv::Mat(VideoFrameH,VideoFrameW,CV_8UC3,hptr2); // 指標變成 cv::Mat
			} 
			t2 = chrono::steady_clock::now();
			//cout << "=================================================================================================" << endl;
			cout << "[" << frameIdx << "] : "  <<  chrono::duration_cast<chrono::milliseconds>(t2-t1).count() << "ms" << endl; 
			



			// 顯示
			cv::imshow("input_frame", input_frame);
			cv::imshow("output_frame",output_frame);
			if(cv::waitKey(1) == 27){ 
		        cout << "Esc !! " << endl; 
		        break; 
		    }//endif
		    frameIdx++;
		}//end_while
		
		// 釋放記憶體
		hipFree(dptr);
		free(hptr2);

	}//endif

	return 0;
}//end_main