/*==========================================================================================

// openvino option :
	"C:\Program Files (x86)\IntelSWTools\openvino\bin\setupvars.bat"


=============================================================================================*/

#include <iostream>
// nvidia cuda library  
#include "npp.h"


// intel-openvino-opencv
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core.hpp>
#include <opencv2/video.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/dnn.hpp>

using namespace std;


int main(int args,char* argv[]){
	if(args != 3){
		cout << "openvino-init : \"C:\\Program Files (x86)\\IntelSWTools\\openvino\\bin\\setupvars.bat\" "
		cout << "--run [videoname]" << endl;
		exit(1);
	}

	// ________________  讀取 .mp4___________________________
	cv::VideoCapture cap(argv[2]);
	int VideoFrameH = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
	int VideoFrameW = cap.get(cv::CAP_PROP_FRAME_WIDTH);
	double fps = cap.get(cv::CAP_PROP_FPS);
	int FrameSize = VideoFrameH * VideoFrameW;
	size_t bytes = FrameSize*3*sizeof(unsigned char);

	

	if(cap.isOpened()){
		//________________ 定義指標_________________________ 
		cv::Mat frame;
		cv::Mat frame2;
		unsigned char* hptr; // CPU 處理前
		unsigned char* dptr; // GPU
		unsigned char* hptr2 = (unsigned char *)malloc(bytes); // CPU 處理後
		hipMalloc(&dptr,bytes);
		
		// ________________ 掃每一禎 ______________________
		while(cap.read(frame)){
			hptr = frame.data;
			hipMemcpy(dptr,hptr,bytes,hipMemcpyHostToDevice);
			// _____________ CUDA HANDLE BLOCK ________________________________________









			//_________________________________________________________________________
			hipMemcpy(hptr2,dptr,bytes,hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			frame2 = cv::Mat(VideoFrameH,VideoFrameW,CV_8UC3,hptr2);

			cv::imshow("frame", frame);
			cv::imshow("frame2",frame2);
			if(cv::waitKey(1) == 27){ 
		        cout << "Esc !! " << endl; 
		        break; 
		    }//endif
		}//end_while

		hipFree(dptr);
		free(hptr2);

	}//endif

	

	return 0;
}//end_main