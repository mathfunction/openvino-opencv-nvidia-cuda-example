#include "hip/hip_runtime.h"
/*==========================================================================================
// 這是關於把一個影片輸入，利用 GPU 運算 ，把彩色影片添加隨機對角線
"C:\\Program Files (x86)\\IntelSWTools\\openvino\\bin\\setupvars.bat"
// opencv == H x W x C (BGR)

=============================================================================================*/
#include <chrono>
#include <iostream>
#include <cstdlib>
// nvidia cuda library  
#include "npp.h"
#include "hipblas.h"
#include "hipsparse.h"
#include <thrust/device_vector.h>

// intel-openvino-opencv
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core.hpp>
#include <opencv2/video.hpp>
#include <opencv2/videoio.hpp>
#include <opencv2/dnn.hpp>
// DIYCUDA
#include "cuda_useful.hpp"


using namespace std;








__global__ void setGPUChannels(
		int *dx,
		int *dy,
		unsigned char *dB,
		unsigned char *dG,
		unsigned char *dR,
		int size,
		int b,
		int g,
		int r){
	__cuda_parallel_1d__(size){
			dx[idx] = idx;
			dy[idx] = idx;
			dB[idx] = (unsigned char)b;
			dG[idx] = (unsigned char)g;
			dR[idx] = (unsigned char)r;
	}
}



void run(const char* filename,int NumBlocks,int NumThreads){
	// ________________  讀取 .mp4___________________________
	cv::VideoCapture cap(filename);                          		 // 輸入影片檔名
	int VideoFrameH = cap.get(cv::CAP_PROP_FRAME_HEIGHT);            // 影片高
	int VideoFrameW = cap.get(cv::CAP_PROP_FRAME_WIDTH);			 // 影片寬
	double fps = cap.get(cv::CAP_PROP_FPS);    						 // 取得該影片 FPS 資訊
	int size = VideoFrameH*VideoFrameW*3;
	size_t bytes = size*sizeof(unsigned char);  // 計算需要傳輸 的 bytes = H x W x C x (uchar) 

	if(cap.isOpened()){
		// 定義指標 + 配置空間
		cv::Mat input_frame;
		cv::Mat output_frame;
		// 對角線 pixel 數
		int smallHW = min(VideoFrameW,VideoFrameH);
		

		// CPU
		unsigned char* hptr; // CPU input_frame
		


		// GPU
		unsigned char* dptr; // GPU
		int* dx;   // GPU x通道
		int* dy;   // GPU y通道
		unsigned char* dB;   // GPU B通道
		unsigned char* dG;   // GPU G通道
		unsigned char* dR;   // GPU R通道

		
		unsigned char* hptr2 = (unsigned char *)malloc(bytes); // CPU output_frame

		hipMalloc(&dptr,bytes);
		hipMalloc(&dx,smallHW*sizeof(int));
		hipMalloc(&dy,smallHW*sizeof(int));
		hipMalloc(&dB,smallHW*sizeof(unsigned char));
		hipMalloc(&dG,smallHW*sizeof(unsigned char));
		hipMalloc(&dR,smallHW*sizeof(unsigned char));
		


		chrono::steady_clock::time_point t1;
		chrono::steady_clock::time_point t2;
		int frameIdx = 1;

	
		
		// ________________ 掃每一禎 ______________________
		while(cap.read(input_frame)){
			int r = rand();
			int g = rand();
			int b = rand();
			t1 = chrono::steady_clock::now();
			{
				hptr = input_frame.data;  //存到指標上


				hipMemcpy(dptr,hptr,bytes,hipMemcpyHostToDevice);
				//===============================================
				// 再 GPU 上配置對角線
				setGPUChannels<<<NumBlocks,NumThreads >>>(dx,dy,dB,dG,dR,smallHW,b,g,r);
				cuda_useful::setBGRs<<< NumBlocks,NumThreads >>>(dptr,dx,dy,VideoFrameW,dB,dG,dR,smallHW);
				//=================================================
				hipMemcpy(hptr2,dptr,bytes,hipMemcpyDeviceToHost);
				hipDeviceSynchronize(); //與主程式同步
				output_frame = cv::Mat(VideoFrameH,VideoFrameW,CV_8UC3,hptr2); // 指標變成 cv::Mat
			} 
			t2 = chrono::steady_clock::now();
			//cout << "=================================================================================================" << endl;
			cout << "f-" << frameIdx <<  " : "  << chrono::duration_cast<chrono::milliseconds>(t2-t1).count() << " ms" <<"["  <<  chrono::duration_cast<chrono::microseconds>(t2-t1).count() << " mus]"  << endl; 
			

			// 顯示
			cv::imshow("input_frame", input_frame);
			cv::imshow("output_frame",output_frame);
			if(cv::waitKey(1) == 27){ 
		        cout << "Esc !! " << endl; 
		        break; 
		    }//endif
		    frameIdx++;
		}//end_while
		
		// 釋放記憶體
		hipFree(dptr);
		free(hptr2);

	}//endif
}




// ctypes code 
extern "C"{
	#define DLLEXPORT __declspec(dllexport)
	DLLEXPORT int cuda_run(char* filename,int NumBlocks,int NumThreads){
		run(filename,NumBlocks,NumThreads);
		return 0;
	}
}


int main(int args,char* argv[]){
	if(args != 5){
		cout << "=====================================================================================\n" ;
		cout << "openvino-init : \"C:\\Program Files (x86)\\IntelSWTools\\openvino\\bin\\setupvars.bat\"\n";
		cout << "=====================================================================================\n" ;
		cout << "--run [videoname] [NumBlocks] [NumThreads]" << endl;
		cout << "=====================================================================================\n";
		exit(1);
	}else{
		if(string(argv[1]) == "--run"){
			run(argv[2],atoi(argv[3]),atoi(argv[4]));
		}//endif
	}
	return 0;
}//end_main





